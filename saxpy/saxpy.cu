#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

// return GB/sec
float GBPerSec(int bytes, float sec)
{
    return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}

// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
saxpy_kernel(int N, float alpha, float *x, float *y, float *result)
{

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    if (index < N)
        result[index] = alpha * x[index] + y[index];
}

// saxpyCuda --
//
// This function is regular C code running on the CPU.  It allocates
// memory on the GPU using CUDA API functions, uses CUDA API functions
// to transfer data from the CPU's memory address space to GPU memory
// address space, and launches the CUDA kernel function on the GPU.
void saxpyCuda(int N, float alpha, float *xarray, float *yarray, float *resultarray)
{

    // must read both input arrays (xarray and yarray) and write to
    // output array (resultarray)
    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block.  In this
    // application we've hardcoded thread blocks to contain 512 CUDA
    // threads.
    const int threadsPerBlock = 512;

    // Notice the round up here.  The code needs to compute the number
    // of threads blocks needed such that there is one thread per
    // element of the arrays.  This code is written to work for values
    // of N that are not multiples of threadPerBlock.
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // These are pointers that will be pointers to memory allocated
    // *one the GPU*.  You should allocate these pointers via
    // hipMalloc.  You can access the resulting buffers from CUDA
    // device kernel code (see the kernel function saxpy_kernel()
    // above) but you cannot access the contents these buffers from
    // this thread. CPU threads cannot issue loads and stores from GPU
    // memory!
    float *device_x = nullptr;
    float *device_y = nullptr;
    float *device_result = nullptr;

    //
    // CS149 TODO: allocate device memory buffers on the GPU using hipMalloc.
    //
    // We highly recommend taking a look at NVIDIA's
    // tutorial, which clearly walks you through the few lines of code
    // you need to write for this part of the assignment:
    //
    // https://devblogs.nvidia.com/easy-introduction-cuda-c-and-c/
    //

    // Allocate device memory
    hipMalloc((void **)&device_x, N * sizeof(float));
    hipMalloc((void **)&device_y, N * sizeof(float));
    hipMalloc((void **)&device_result, N * sizeof(float));

    // Copy input arrays to the GPU
    hipMemcpy(device_x, xarray, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, N * sizeof(float), hipMemcpyHostToDevice);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // CS149 TODO: copy input arrays to the GPU using hipMemcpy
    //

    // Kernel invocation with timing
    double kernelStartTime = CycleTimer::currentSeconds();

    // run CUDA kernel. (notice the <<< >>> brackets indicating a CUDA
    // kernel launch) Execution on the GPU occurs here.
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);

    //
    // CS149 TODO: copy result from GPU back to CPU using hipMemcpy
    //

    hipDeviceSynchronize();
    double kernelEndTime = CycleTimer::currentSeconds();
    double kernelDuration = kernelEndTime - kernelStartTime;
    printf("Kernel execution time: %.3f ms\n", 1000.f * kernelDuration);

    // Copy result from GPU back to CPU
    hipMemcpy(resultarray, device_result, N * sizeof(float), hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();


    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess)
    {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
                errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Effective BW by CUDA saxpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, GBPerSec(totalBytes, overallDuration));

    //
    // CS149 TODO: free memory buffers on the GPU using hipFree
    //
    // Free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void printCudaInfo()
{

    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

// Question 1. What performance do you observe compared to the sequential
// CPU-based implementation of SAXPY (recall your results from saxpy on Program
// 5 from Assignment 1)?

// One would need to run both the CPU-based implementation and their new CUDA-based
// implementation to observe and compare their performance. Typically, the CUDA implementation
// is found to be significantly faster for large arrays due to parallel processing on the GPU.

// Question 2. Compare and explain the difference between the results provided
// by two sets of timers (timing only the kernel execution vs. timing the entire
// process of moving data to the GPU and back in addition to the kernel
// execution). Are the bandwidth values observed roughly consistent with the
// reported bandwidths available to the different components of the machine?
// (You should use the web to track down the memory bandwidth of an NVIDIA T4
// GPU. Hint:
// https://www.nvidia.com/content/dam/en-zz/Solutions/Data-Center/tesla-t4/t4-tensor-core-datasheet-951643.pdf.
// The expected bandwidth of memory bus of AWS is 4 GB/s, which does not match
// that of a 16-lane PCIe 3.0. Several factors prevent peak bandwidth, including
// CPU motherboard chipset performance and whether or not the host CPU memory
// used as the source of the transfer is “pinned” — the latter allows the GPU to
// directly access memory without going through virtual memory address
// translation. If you are interested, you can find more info here:
// https://kth.instructure.com/courses/12406/pages/optimizing-host-device-data-communication-i-pinned-host-memory)

// The difference in results between timing only the kernel execution vs. timing the entire process (including data
// transfer to and from the GPU) is crucial. Data transfer over PCIe can be a bottleneck due to its lower bandwidth
// compared to GPU memory bandwidth. The observed bandwidth values for kernel execution should be higher and more
// consistent with the GPU's memory bandwidth. In contrast, the overall process including data transfer might align
// more closely with PCIe bandwidth limitations, affecting total performance.